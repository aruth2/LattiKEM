#include "hip/hip_runtime.h"
#include "bandgap_GPU.h"
int GPU_threads;
int sDATA_SZ;

__device__ void fermiIteration(double *energyLevels, double fermiGuess, double temperature, int numEnergyLevels, double *weights, double *energyWeights, double *weightSum)
{
	double threadEnergy=0;//Sum for each thread
	double threadWeight=0;
	int iLevel; 
	for(iLevel = threadIdx.x;iLevel < numEnergyLevels;iLevel+=blockDim.x)
	{
		weights[iLevel] = 1.0/(1.0+exp((*(energyLevels+iLevel)-fermiGuess)/temperature));
		threadEnergy +=weights[iLevel] * energyLevels[iLevel];
		threadWeight += weights[iLevel];
	}

	energyWeights[threadIdx.x] = threadEnergy; //Store sum from each thread at base value
	weightSum[threadIdx.x] = threadWeight; //Store sum from each thread at base value
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) 
	{
		__syncthreads();

		if(threadIdx.x < stride)
		{
			weightSum[threadIdx.x ] += weightSum[threadIdx.x+stride];
			energyWeights[threadIdx.x] += energyWeights[threadIdx.x+stride];
		}
	}
	__syncthreads();
}

__global__ void fermiKernel(double *energyLevels, double *fermiEnergy, double currentExcitations, double fermiConvergence, double temperature, int numEnergyLevels,  double *weights, double *energyWeights, double *d_energy, double *weightSum)
{
	//printf("Running kernel on GPU\n");
	double fermiGuess = *fermiEnergy;
	double fermiMin = fermiGuess - 7.5*fermiConvergence;
	double fermiMax = fermiGuess + 8.49*fermiConvergence;//This is slightly offset so that the fermi level could be almost unchanged. 

	double totalCarriersMin=0;
	double totalCarriersMax=0; 
	double totalCarriers;
	
	//Try to bound the fermi level within a small box around the previous result which requires not more than 6 iterations to converge.
	fermiIteration(energyLevels, fermiMin, temperature, numEnergyLevels, weights, energyWeights, weightSum);
	totalCarriersMin = *weightSum;
	fermiIteration(energyLevels, fermiMax, temperature, numEnergyLevels, weights, energyWeights, weightSum);
	totalCarriersMax = *weightSum;
	
	if((totalCarriersMin > currentExcitations) || (totalCarriersMax < currentExcitations))//If the solution is not within the bounds expand the bounds.  	
	{
		//printf("Fermi level outside bounds, Min %.10g max %.10g, min carriers %.10g, max carriers %.10g, target carriers %g\n",fermiMin,fermiMax,totalCarriersMin,totalCarriersMax,currentExcitations);
		fermiMin = optics_llimit;
		fermiMax = optics_ulimit;
	}
	
	while(fermiMax - fermiMin > fermiConvergence)
	{
 
		fermiGuess = (fermiMax+fermiMin)/2.0;	
		
		__syncthreads();
		fermiIteration(energyLevels, fermiGuess, temperature, numEnergyLevels, weights, energyWeights, weightSum);
		totalCarriers = *weightSum;
	
		if(totalCarriers > currentExcitations) //Lower search range
			fermiMax = fermiGuess;
		else
			fermiMin = fermiGuess;
		//printf("Fermi range %g %g Carriers %g %g\n",fermiMin,fermiMax,totalCarriers,currentExcitations);__syncthreads();
	}
	
	if(threadIdx.x == 0)
	{
		*fermiEnergy = fermiGuess;
		*d_energy = *energyWeights;
	}
}

__host__ void bgg_runKernel(OptoelectronicState *OS, int copyWeights, int num_states, double currentExcitations)
{
	double temperature = getTemperature();
	
	nvtxRangePushA(":ELEVELS_IN");
	hipMemcpyAsync(OS->d_energyLevels, OS->energystates, sDATA_SZ, hipMemcpyHostToDevice,0);
	//hipStreamSynchronize(0);
	nvtxRangePop();
	
	nvtxRangePushA(":KERNEL");	
	fermiKernel<<<1, GPU_threads,0,0>>>(OS->d_energyLevels, OS->d_fermiEnergy,  currentExcitations, getFermiConvergence() , temperature, num_states, OS->d_weights,OS->d_energyWeights,OS->d_energy,OS->d_weightSum);
 	//hipStreamSynchronize(0);		
	nvtxRangePop();
	
	if(copyWeights)
	{
	nvtxRangePushA(":WEIGHTS_OUT");
	hipMemcpyAsync(OS->weights, OS->d_weights, sDATA_SZ, hipMemcpyDeviceToHost,0);
	//hipStreamSynchronize(0);
	nvtxRangePop();
	}
	
	nvtxRangePushA(":FENERGY_OUT");
	hipMemcpyAsync(OS->fermiEnergy, OS->d_fermiEnergy,  sizeof(double), hipMemcpyDeviceToHost,0);
	//hipStreamSynchronize(0); 
	nvtxRangePop();
	
	nvtxRangePushA( ":ENERGY_OUT");
	hipMemcpyAsync(OS->photocarrierEnergy, OS->d_energy, sizeof(double), hipMemcpyDeviceToHost,0);
	hipStreamSynchronize(0);
	nvtxRangePop(); 
}

__host__ void bindGPU(int threadnumber)
{
	if(threadnumber == -1)
	return;
	
	int num_gpu;
	hipGetDeviceCount(&num_gpu);
	int iGPU = num_gpu * thread_affinity(threadnumber);
	hipSetDevice(iGPU);
}

__host__ void bgg_allocate(Configuration *config, int num_states)
{
	//printf("Allocating on GPU\n");
	OptoelectronicState *OS = (OptoelectronicState *)(config->data);
	bindGPU(config->threadnumber);
	
	int saccum_size = GPU_threads*sizeof(double);
	sDATA_SZ = num_states * sizeof(double);
	nvtxRangePushA("Allocate");
	hipMalloc((void **)&(OS->d_energyLevels), sDATA_SZ);
	hipMalloc((void **)&(OS->d_weights), sDATA_SZ);
	hipMalloc((void **)&(OS->d_energyWeights), saccum_size);
	hipMalloc((void **)&(OS->d_energy), sizeof(double));
	hipMalloc((void **)&(OS->d_weightSum), saccum_size);
	hipMalloc((void **)&(OS->d_fermiEnergy), sizeof(double));
	
	hipHostMalloc((void**)&(OS->energystates), sDATA_SZ);
	hipHostMalloc((void**)&(OS->weights), sDATA_SZ);
	hipHostMalloc((void**)&(OS->photocarrierEnergy), sizeof(double));
	hipHostMalloc((void**)&(OS->fermiEnergy), sizeof(double));
	
	hipStreamSynchronize(0);//Finish Mallocs before starting Memcpy
  	nvtxRangePop();
}

__host__ void bgg_free(OptoelectronicState *OS)
{
	hipFree(OS->d_energyLevels);
	hipFree(OS->d_weights);
	hipFree(OS->d_energyWeights);
	hipFree(OS->d_energy);
	hipFree(OS->d_weightSum);
	hipFree(OS->d_fermiEnergy);
	
	hipFree(OS->energystates);
	hipFree(OS->weights);
	hipFree(OS->photocarrierEnergy);
	hipFree(OS->fermiEnergy);
	
}

__host__ void bgg_registerSettings()
{
	registerInt(&GPU_threads,"GPU_threads",1024);
	//registerDouble(&fermiConvergence,"fermiConvergence",1e-5);
}
