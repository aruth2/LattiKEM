//#include "distribution_GPU.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <pthread.h>
#include <hip/hip_runtime.h>

#include <nvtx3/nvToolsExt.h>
#include <numa.h>
enum MEMCPY_BITMASK {ELEVELS_IN, FENERGY_IN, WEIGHTS_OUT, FENERGY_OUT, ENERGY_OUT};

#define optics_llimit -1.0
#define optics_ulimit 1.0
__device__ void fermiIteration(double *energyLevels, double fermiGuess, double temperature, int numEnergyLevels, double *weights, double *energyWeights, double *weightSum)
{
	double threadEnergy=0;//Sum for each thread
	double threadWeight=0;
	int iLevel; 
	for(iLevel = threadIdx.x;iLevel < numEnergyLevels;iLevel+=blockDim.x)
	{
		weights[iLevel] = 1.0/(1.0+exp((*(energyLevels+iLevel)-fermiGuess)/temperature));
		threadEnergy +=weights[iLevel] * energyLevels[iLevel];
		threadWeight += weights[iLevel];
	}

	energyWeights[threadIdx.x] = threadEnergy; //Store sum from each thread at base value
	weightSum[threadIdx.x] = threadWeight; //Store sum from each thread at base value
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) 
	{
		__syncthreads();

		if(threadIdx.x < stride)
		{
			weightSum[threadIdx.x ] += weightSum[threadIdx.x+stride];
			energyWeights[threadIdx.x] += energyWeights[threadIdx.x+stride];
		}
	}
	__syncthreads();
}

__global__ void fermiKernel(double *energyLevels, double *fermiEnergy, double currentExcitations, double fermiConvergence, double temperature, int numEnergyLevels,  double *weights, double *energyWeights, double *d_energy, double *weightSum)
{
	//double fermiGuess = 0.5;
	double fermiGuess = *fermiEnergy;
	double fermiMin = fermiGuess - 7.5*fermiConvergence;
	double fermiMax = fermiGuess + 8.49*fermiConvergence;//This is slightly offset so that the fermi level could be almost unchanged. 
	//double fermiEnergy;
	double totalCarriersMin=0;
	double totalCarriersMax=0; 
	double totalCarriers;
	//Try to bound the fermi level within a small box around the previous result which requires not more than 6 iterations to converge.
	fermiIteration(energyLevels, fermiMin, temperature, numEnergyLevels, weights, energyWeights, weightSum);
	//fermiIterationMultiblock(energyLevels, fermiMin, temperature, numEnergyLevels, weights, energyWeights, weightSum);
	totalCarriersMin = *weightSum;
	fermiIteration(energyLevels, fermiMax, temperature, numEnergyLevels, weights, energyWeights, weightSum);
	//fermiIterationMultiblock(energyLevels, fermiMax, temperature, numEnergyLevels, weights, energyWeights, weightSum);
	totalCarriersMax = *weightSum;
	
	if((totalCarriersMin > currentExcitations) || (totalCarriersMax < currentExcitations))//If the solution is not within the bounds expand the bounds.  	
	{
		//printf("Fermi level outside bounds, Min %.10g max %.10g, min carriers %.10g, max carriers %.10g, target carriers %g\n",fermiMin,fermiMax,totalCarriersMin,totalCarriersMax,currentExcitations);
		fermiMin = optics_llimit;
		fermiMax = optics_ulimit;
	}
	
	while(fermiMax - fermiMin > fermiConvergence)
	{
 
		fermiGuess = (fermiMax+fermiMin)/2.0;	
		
		__syncthreads();
		fermiIteration(energyLevels, fermiGuess, temperature, numEnergyLevels, weights, energyWeights, weightSum);
		//fermiIterationMultiblock(energyLevels, fermiGuess, temperature, numEnergyLevels, weights, energyWeights, weightSum);
		totalCarriers = *weightSum;
	
		if(totalCarriers > currentExcitations) //Lower search range
			fermiMax = fermiGuess;
		else
			fermiMin = fermiGuess;
		//printf("Fermi range %g %g Carriers %g %g\n",fermiMin,fermiMax,totalCarriers,currentExcitations);__syncthreads();
	}
	
	if(threadIdx.x == 0)
	{
		*fermiEnergy = fermiGuess;
		*d_energy = *energyWeights;
	}
}


void fermiSum( double *energyLevels, double fermiEnergy, double temperature, int numEnergyLevels, double *weights, double *weightSum, double *energySum)
{
 	/*
	Evaluates the fermi Sum, N=sum(fi) and E=sum(fi * Ei)
	where fi is the fermi function fi = 1/(1+exp((Ei-Ef)/kT))
	
	Inputs are:
	energyLevels (Ei)
	fermiEnergy (Ef)
	temperature (kT)
	numEnergyLevels (n)
	
	outputs are:
	weights (fi)
	weightSum (N)
	energySum (E)
	*/
	*weightSum=0;
	*energySum=0;
	for(int i = 0;i<numEnergyLevels;i++)
	{
		*(weights+i) = 1.0/(1.0+exp((*(energyLevels+i)-fermiEnergy)/temperature));
		*weightSum += *(weights+i);
		*energySum += *(energyLevels+i) * *(weights+i);
	}
}


double solveFermiEnergy(double currentExcitations, double fermiGuess, double fermiConvergence, double *energyLevels,double *weights, int numEnergyLevels, double temperature, double *energy)
{
	double fermiMin = fermiGuess - 7.5*fermiConvergence;
	double fermiMax = fermiGuess + 8.49*fermiConvergence;//This is slightly offset so that the fermi level could be almost unchanged. 
	//double fermiEnergy;
	double totalCarriersMin=0;
	double totalCarriersMax=0; 
	//Try to bound the fermi level within a small box around the previous result which requires not more than 6 iterations to converge.
	fermiSum( energyLevels, fermiMin, temperature, numEnergyLevels, weights, &totalCarriersMin, energy);
	fermiSum( energyLevels, fermiMax, temperature, numEnergyLevels, weights, &totalCarriersMax, energy);
	
	
	if((totalCarriersMin > currentExcitations) || (totalCarriersMax < currentExcitations))//If the solution is not within the bounds expand the bounds.  	
	{
		//printf("Fermi level outside bounds, Min %.10g max %.10g, min carriers %.10g, max carriers %.10g, target carriers %g\n",fermiMin,fermiMax,totalCarriersMin,totalCarriersMax,currentExcitations);
		fermiMin = optics_llimit;
		fermiMax = optics_ulimit;
	}
		
	double totalCarriers=0;
	while(fermiMax-fermiMin > fermiConvergence)
	{
		fermiGuess = (fermiMax+fermiMin)/2.0;	
		fermiSum( energyLevels, fermiGuess, temperature, numEnergyLevels, weights, &totalCarriers, energy);
		if(totalCarriers > currentExcitations) //Lower search range
			fermiMax = fermiGuess;
		else
			fermiMin = fermiGuess;
		//printf("Fermi range %g %g Carriers %g %g\n",fermiMin,fermiMax,totalCarriers,currentExcitations);
	}
	return fermiGuess;
}

double RandDouble(double low, double high, unsigned int *state) {
	double t = (double)rand_r(state) / (double)RAND_MAX;
	return (1.0f - t) * low + t * high;
}

//https://stackoverflow.com/questions/1407786/how-to-set-cpu-affinity-of-a-particular-pthread
// core_id = 0, 1, ... n-1, where n is the system's number of cores

int stick_this_thread_to_core(int core_id) {
   int num_cores = sysconf(_SC_NPROCESSORS_ONLN);
   if (core_id < 0 || core_id >= num_cores)
      return EINVAL;

   cpu_set_t cpuset;
   CPU_ZERO(&cpuset);
   CPU_SET(core_id, &cpuset);

   pthread_t current_thread = pthread_self();    
   return pthread_setaffinity_np(current_thread, sizeof(cpu_set_t), &cpuset);
}

// Total number of input vector pairs; 
//const int num_pthreads = 256;
int num_pthreads = 256;
// Number of elements per vector; arbitrary,
// but strongly preferred to be a multiple of warp size
// to meet memory coalescing constraints
int ELEMENT_N = 64000;
//const int ELEMENT_N = 64000;
// Total number of data elements
int sDATA_SZ;
//const int sDATA_SZ = ELEMENT_N * sizeof(double);
int num_blocks = 1;
int num_threads = 128;
int run_mask = 3;
int repeats = 1;
int memcpy_mask = 31;
double fermiConvergence = 1e-7;

#define hyperthreads 2
#define sockets 1
#define nodesPerSocket 4	
void * single_thread(void *data)
{
	int thread_number = *((int *)data);
	stick_this_thread_to_core(thread_number);
	int num_gpu;
	hipGetDeviceCount(&num_gpu);
	//int iNUMA = (double)(thread_number % 64)/16;
	int cores = num_pthreads/hyperthreads;
	int iNUMA = sockets * nodesPerSocket * ((double)(thread_number % cores))/(cores);
	int iGPU = num_gpu * ((double)(thread_number % cores))/(cores);
	//printf("Thread %d cores %d nodesPerSocket %d NUMA %d\n",thread_number,cores,nodesPerSocket,iNUMA);
	//cudaSetDevice((double)thread_number/num_threads*num_gpu);
	//cudaSetDevice(3-iNUMA);
	hipSetDevice(iGPU);
	double  *h_weights, *weights_CPU, fermiEnergy_CPU,  energy_CPU;
	double *h_energy, *h_fermiEnergy;
	double *h_energyLevels;
	double *d_energyLevels, *d_weights, *d_energyWeights, *d_weightSum, *d_fermiEnergy, *d_energy;
	int i,iRepeat,ielevel;
	int num_elevel_update = 16;
	int *elevel_update = (int *)malloc(num_elevel_update*sizeof(int));	
	int saccum_size = num_threads*sizeof(double);
	if(thread_number==0)
		printf("Initializing data...\n");
	if(thread_number==0)
		printf("...allocating CPU memory.\n");
	
	//cudaMallocManaged((void**)&h_energyLevels, sDATA_SZ);
	hipHostMalloc((void**)&h_energyLevels, sDATA_SZ, hipHostMallocDefault);
	//h_energyLevels = (double *)malloc(sDATA_SZ);
	//h_energyLevels = (double *)numa_alloc_onnode(sDATA_SZ,iNUMA);
	//cudaMallocManaged((void**)&h_weights, sDATA_SZ);
	hipHostMalloc((void**)&h_weights, sDATA_SZ, hipHostMallocDefault);
	//h_weights = (double *)malloc(sDATA_SZ);
	//h_weights = (double *)numa_alloc_onnode(sDATA_SZ,iNUMA);
	//cudaMallocManaged((void**)&h_energy, sizeof(double));
	hipHostMalloc((void**)&h_energy, sizeof(double), hipHostMallocDefault);
	//h_energy = (double *)malloc(sizeof(double));
	//h_energy = (double *)numa_alloc_onnode(sizeof(double),iNUMA);
	//cudaMallocManaged((void**)&h_fermiEnergy, sizeof(double));
	hipHostMalloc((void**)&h_fermiEnergy, sizeof(double), hipHostMallocDefault);
	//h_fermiEnergy = (double *)malloc(sizeof(double));
	//h_fermiEnergy = (double *)numa_alloc_onnode(sizeof(double),iNUMA);
	weights_CPU = (double *)malloc(sDATA_SZ);

	if(thread_number==0)
		printf("...allocating GPU memory.\n");
	
	nvtxRangePushA(":CUDAMalloc");
	hipMalloc((void **)&d_energyLevels, sDATA_SZ);
	hipMalloc((void **)&d_weights, sDATA_SZ);
	hipMalloc((void **)&d_energyWeights, saccum_size);
	hipMalloc((void **)&d_energy, sizeof(double));
	hipMalloc((void **)&d_weightSum, saccum_size);
	hipMalloc((void **)&d_fermiEnergy, sizeof(double));
	
  	hipStreamSynchronize(0);//Finish Mallocs before starting Memcpy
  	nvtxRangePop();
  	
  	if(thread_number==0)
		printf("...initializing data.\n");
	
	unsigned int state;	
	for (i = 0; i < ELEMENT_N; i++) 
	{
		h_energyLevels[i] = RandDouble(0.0f, 1.0f,&state);
	}
	*h_fermiEnergy = 0.5;
	
  
	if((run_mask / 2) % 2 == 1)
	{
		if(thread_number==0)
			printf("...Beginning CPU calculation.\n");
		for(iRepeat=0;iRepeat<repeats;iRepeat++)
		{
			fermiEnergy_CPU =  solveFermiEnergy( ELEMENT_N/2, fermiEnergy_CPU, fermiConvergence, h_energyLevels,weights_CPU,ELEMENT_N,0.025,&energy_CPU);
		
		}
		if(thread_number==0)	
			printf("Fermi Level from CPU is %g energy %g\n",fermiEnergy_CPU,energy_CPU); 
	}

	if ((run_mask % 2 == 1))
	{
		if(thread_number==0)
		printf("...copying input data to GPU mem.\n");
		
		// Copy options data to GPU memory for further processing
		hipMemcpyAsync(d_energyLevels, h_energyLevels, sDATA_SZ, hipMemcpyHostToDevice,0);
		//cudaMemcpyAsync(d_fermiEnergy, h_fermiEnergy, sizeof(double), cudaMemcpyHostToDevice,0);
  		hipStreamSynchronize(0);
  		
  		if(thread_number==0)
		printf("...Running GPU Kernel.\n");
			
		for(iRepeat=0;iRepeat<repeats;iRepeat++)
		{
			if((memcpy_mask >> ELEVELS_IN) %2)
			{
				//Generate random list of updates outside NVTX timing
				for(ielevel=0;ielevel<num_elevel_update;ielevel++)
					elevel_update[ielevel]=RandDouble(0,1,&state);
				nvtxRangePushA(":ELEVELS_IN");
				//for(ielevel=0;ielevel<num_elevel_update;ielevel++) 
					//cudaMemcpyAsync(d_energyLevels+elevel_update[ielevel], h_energyLevels+elevel_update[ielevel], sizeof(double), cudaMemcpyHostToDevice,0);
				hipMemcpyAsync(d_energyLevels, h_energyLevels, sDATA_SZ, hipMemcpyHostToDevice,0);
				//cudaMemcpyAsync(d_energyLevels, h_energyLevels, 16*sizeof(double), cudaMemcpyHostToDevice,0);
				hipStreamSynchronize(0);
				nvtxRangePop();
			}
			
			if((memcpy_mask >> FENERGY_IN) %2)
			{
				nvtxRangePushA(":FENERGY_IN"); 
				hipMemcpyAsync(d_fermiEnergy, h_fermiEnergy, sizeof(double), hipMemcpyHostToDevice,0);
				hipStreamSynchronize(0);
				nvtxRangePop();
			}
			nvtxRangePushA(":KERNEL");	
			fermiKernel<<<1, num_threads,0,0>>>(d_energyLevels, d_fermiEnergy,  ELEMENT_N/2, fermiConvergence , 0.025, ELEMENT_N, d_weights,d_energyWeights,d_energy,d_weightSum);
 			hipStreamSynchronize(0);		
			nvtxRangePop();
				
			
			if((memcpy_mask >> WEIGHTS_OUT) %2)
			{
				nvtxRangePushA(":WEIGHTS_OUT");
				hipMemcpyAsync(h_weights, d_weights, sDATA_SZ, hipMemcpyDeviceToHost,0);
				hipStreamSynchronize(0);
				nvtxRangePop();
			}
			
			if((memcpy_mask >> FENERGY_OUT) %2)
			{
				nvtxRangePushA(":FENERGY_OUT");
				hipMemcpyAsync(h_fermiEnergy, d_fermiEnergy,  sizeof(double), hipMemcpyDeviceToHost,0);
				hipStreamSynchronize(0); 
				nvtxRangePop();
			}
			
			if((memcpy_mask >> ENERGY_OUT) %2)
			{ 
				nvtxRangePushA( ":ENERGY_OUT");
				hipMemcpyAsync(h_energy, d_energy, sizeof(double), hipMemcpyDeviceToHost,0);
				hipStreamSynchronize(0);
				nvtxRangePop(); 
			}
  		}

  		hipMemcpyAsync(h_weights, d_weights, sDATA_SZ, hipMemcpyDeviceToHost,0);
		hipMemcpyAsync(h_fermiEnergy, d_fermiEnergy,  sizeof(double), hipMemcpyDeviceToHost,0);  
		hipMemcpyAsync(h_energy, d_energy, sizeof(double), hipMemcpyDeviceToHost,0); 	
  		
  		hipStreamSynchronize(0);	
 
  		//cudaDeviceSynchronize();	
  		if(thread_number==0)
  		printf("Fermi Level GPU is %g Energy GPU is %g\n",*h_fermiEnergy,*h_energy);
  		//printf("Fermi Level GPU is %g Energy GPU is %g\n",*d_fermiEnergy,*d_energy);
  		hipStreamSynchronize(0);//Finish Memcpys before Cuda_free
  	}
  	
	if(thread_number==0)
  	printf("...freeing GPU memory.\n");
  	nvtxRangePushA(":CUDAFree");
	hipFree(d_energyLevels);
	hipFree(d_weights);
	hipFree(d_energyWeights);
	hipFree(d_weightSum);
	hipFree(d_fermiEnergy);
	hipFree(d_energy);
	hipStreamSynchronize(0);
	nvtxRangePop(); 
	return 0;
}

int main(int argc, char **argv)
{

	if (argc == 8)
	{
		num_pthreads = atoi(argv[1]);
		num_blocks = atoi(argv[2]);
		num_threads = atoi(argv[3]);
		run_mask = atoi(argv[4]);
		repeats = atoi(argv[5]);
		ELEMENT_N = atoi(argv[6]);
		memcpy_mask = atoi(argv[7]);
	}
	else
		printf("Usage: distribution_GPU  num_pthreads num_blocks num_threads run_mask repeats elements memcpy_mask[0-31]\n");
	
	
	printf("Running with %d pthreads %d blocks %d threads, %d run_mask and %d repeats %d elements\n",num_pthreads,num_blocks,num_threads,run_mask,repeats,ELEMENT_N);
	if((memcpy_mask >> ELEVELS_IN) %2)
		printf("Copying energy levels in\n");
	if((memcpy_mask >> FENERGY_IN) %2)
		printf("Copying fermi energy in\n");
	if((memcpy_mask >> WEIGHTS_OUT) %2)
		printf("Copying weights out\n");
	if((memcpy_mask >> FENERGY_OUT) %2)
		printf("Copying fermi energy out\n");
	if((memcpy_mask >> ENERGY_OUT) %2)
		printf("Copying energy out\n");
	printf("%s Starting...\n\n", argv[0]);
	sDATA_SZ = ELEMENT_N * sizeof(double);

  	int iThread = 0;
  	int *threadNumber = (int *)malloc(num_pthreads*sizeof(int));
  	void *retval;
  	pthread_t *threads = (pthread_t *)malloc(num_pthreads*sizeof(pthread_t));
  	for(iThread=0;iThread<num_pthreads;iThread++)
  	{
  		*(threadNumber+iThread) = iThread;
  		pthread_create(threads+iThread,NULL,(void *(*)(void *))single_thread,(void *)(threadNumber+iThread));
  		//single_thread((void *)(&iThread));
  	}
  	for(iThread=0;iThread<num_pthreads;iThread++)
  	{
  		pthread_join(*(threads+iThread),&retval);
  	}
}


